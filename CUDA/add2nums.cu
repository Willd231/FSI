
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void AddIntsCUDA(int * a, int * b)
{
a[0] += b[0];
}


int main (){

int a = 5, b = 9;

int *d_a, *d_b;

//then space for the data must be allocated on the gpu 

hipMalloc(&d_a, sizeof(int));
hipMalloc(&d_b, sizeof(int));

//and then go ahead and copy the actual data to the DEVICE once the memory is allocated

hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice); 

// and then call the kernel to actually perform the calculation once you have the memory allocated for the data and it copied over

AddIntsCUDA<<<1,1>>> (d_a, d_b); 


//following the calculation you must then copy the data back to the HOST 

hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

cout <<"The answer is "<<a<<endl;

return 0; 
} 
