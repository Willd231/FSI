#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <ctype.h>
#include <stdint.h>            // for int16_t
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#define MAX_THREADS        1
#define BYTES_PER_SAMPLE   2

#define CUDA_CHECK(call)                                                        
    do {                                                                        
        hipError_t _err = (call);                                              
        if (_err != hipSuccess) {    
            printf("CUDA Error %s:%d: %s\n",                       
                    __FILE__, __LINE__, hipGetErrorString(_err));                                          
            fprintf(stderr, "CUDA Error %s:%d: %s\n",                       
                    __FILE__, __LINE__, hipGetErrorString(_err));              
            exit(EXIT_FAILURE);                                                 
        }                                                                       
    } while (0)

#define CUFFT_CHECK(call)                                                       
    do {                                                                        
        hipfftResult _status = (call);                                           
        if (_status != HIPFFT_SUCCESS) {      
            printf("CUFFT Error %s:%d: %d\n",                      
                    __FILE__, __LINE__, (int)_status);                                   
            fprintf(stderr, "cuFFT Error %s:%d: %d\n",                      
                    __FILE__, __LINE__, (int)_status);                         
            exit(EXIT_FAILURE);                                                 
        }                                                                       
    } while (0)
    
typedef struct {
    hipfftHandle *plans;
    int          doneplan;
} cufftplan;

typedef struct {
    int            init;
    int            ntoread;
    unsigned char *buffer;
} datareader_t;

/* Global parameters */
int  nchan        = 128;
int  ninp         = 4;
int  debug        = 0;
int  naver        = 99999;
int  prod_type    = 'B';
char *infilename  = NULL;
char *outfilename = NULL;
int  nskip        = 0;

/* Function prototypes */
void   print_usage(char *const argv[]);
void   parse_cmdline(int argc, char *const argv[], const char *optstring);
void   openFiles(const char *infile, const char *outfile, int prod_type, FILE **fin, FILE **fout_ac, FILE **fout_cc);
float  elapsed_time(struct timeval *start);
int    readData(datareader_t *r, int nchan, int ninp, FILE *fpin, hipfftComplex **inp_buf);
void   do_FFT_fftw(cufftplan *plan, int nchan, int ninp, hipfftComplex **inp_buf, hipfftComplex **ft_buf);
void   do_CMAC(const int nchan, const int ninp, const int prod_type, hipfftComplex **ft_buf, hipfftComplex **corr_buf);
void   writeOutput(FILE *fout_ac, FILE *fout_cc, int ninp, int nchan, int iter, int prod_type, hipfftComplex **buf, float normaliser);

int main(int argc, char *const argv[]) {
    int             filedone = 0;
    int             res = 0;
    int             iter = 0;
    int             nav_written = 0;
    int             ncorr = ninp * (ninp + 1) / 2;
    FILE           *finp = NULL, *fout_ac = NULL, *fout_cc = NULL;
    float           normaliser = 1.0f;
    float           read_time = 0, fft_time = 0, write_time = 0;
    hipfftComplex  **inp_buf = NULL, **ft_buf = NULL;
    struct timeval  tv;
    cufftplan       fftplan[MAX_THREADS];
    datareader_t    reader[MAX_THREADS];
    long long       loop_count = 0;

    // Initialize plan and reader structures
    for (int t = 0; t < MAX_THREADS; t++) {
        fftplan[t].doneplan = 0;
        fftplan[t].plans    = NULL;
        reader[t].init      = 0;
        reader[t].buffer    = NULL;
    }

    // Parse command line
    if (argc < 2) print_usage((char *const *)argv);
    parse_cmdline(argc, (char *const *)argv, "dc:i:o:n:a:p:s:");

    if (!infilename || !outfilename) {
        fprintf(stderr, "Error: must specify -i and -o\n");
        print_usage((char *const *)argv);
    }

    openFiles(infilename, outfilename, prod_type,
              &finp, &fout_ac, &fout_cc);

    // Initialize CUDA
    CUDA_CHECK(hipFree(0));
    if (debug) fprintf(stderr, "CUDA runtime init\n");

    // Allocate host-managed buffers
    inp_buf = (hipfftComplex**)calloc(ninp, sizeof(hipfftComplex*));
    ft_buf = (hipfftComplex**)calloc(ncorr, sizeof(hipfftComplex*));
    if (!inp_buf || !ft_buf) { perror("calloc"); exit(EXIT_FAILURE); }

    for (int i = 0; i < ninp; i++) {
        CUDA_CHECK(hipMallocManaged((void**)&inp_buf[i], nchan * sizeof(hipfftComplex)));
    }

    for (int i = 0; i < ncorr; i++) {
        CUDA_CHECK(hipMallocManaged((void**)&ft_buf[i], nchan * sizeof(hipfftComplex)));
        if (!ft_buf[i]) {
            fprintf(stderr, "Error: hipMallocManaged failed for ft_buf[%d]\n", i);
            exit(EXIT_FAILURE);
        }
    }

    // Skip initial spectra if needed
    for (int i = 0; i < nskip; i++) {
        readData(&reader[0], nchan, ninp, finp, inp_buf);
    }

    /* process file */
    while (!filedone) {

        /* read time chunk into buffers */
        gettimeofday(&tv, NULL);
        res = readData(&reader[0], nchan, ninp, finp, inp_buf);
        if (debug && res)
            fprintf(stderr, "EOF reached @ iteration %lld\n", loop_count);
        read_time += elapsed_time(&tv);
        if (res) filedone = 1;

        if (!filedone) {
            if (debug && (loop_count % 1000) == 0)
                fprintf(stderr, "db  fft iteration %lld\n", loop_count);

            /* do the FFT */
            gettimeofday(&tv, NULL);
            do_FFT_fftw(&fftplan[0], nchan, ninp, inp_buf, ft_buf);
            CUDA_CHECK(hipDeviceSynchronize());
            fft_time += elapsed_time(&tv);

            /* do the CMAC */
            gettimeofday(&tv, NULL);
            do_CMAC(nchan, ninp, prod_type, ft_buf, ft_buf);
            CUDA_CHECK(hipDeviceSynchronize());
            cmac_time += elapsed_time(&tv);


        }

        /* write and average if it is time to */
        if ((filedone && !nav_written) || ++iter == naver) {
            if (debug) fprintf(stderr, "Calling writeOutput: iter=%d, filedone=%d\n", iter, filedone);
            gettimeofday(&tv, NULL);
            normaliser = 1.0f / (nchan * iter);
            writeOutput(fout_ac, fout_cc, ninp, nchan, iter, prod_type, ft_buf, normaliser);

            //may need to remove this
            if (fout_ac) fflush(fout_ac);
            if (fout_cc) fflush(fout_cc);
            //---------------

            nav_written++;
            iter = 0;
            write_time += elapsed_time(&tv);
        }

        loop_count++;
    }

    // Final debug summary
    if (debug) {
        fprintf(stderr,
                "read=%.3fms  fft=%.3fms  write=%.3fms\n",
                read_time, fft_time, write_time);
    }

    /* clean up */
    if (finp && finp != stdin) fclose(finp);
    if (fout_ac && fout_ac != stdout) fclose(fout_ac);
    if (fout_cc && fout_cc != stdout) fclose(fout_cc);

    for (int i = 0; i < ninp; i++) {
        hipFree(inp_buf[i]);
    }
    for (int i = 0; i < ncorr; i++) {
        hipFree(ft_buf[i]);
    }

    free(inp_buf);
    free(ft_buf);

    for (int t = 0; t < MAX_THREADS; t++) {
        if (fftplan[t].doneplan) {
            for (int j = 0; j < ninp; j++) {
                CUFFT_CHECK(hipfftDestroy(fftplan[t].plans[j]));
            }
            free(fftplan[t].plans);
        }
    }

    for (int t = 0; t < MAX_THREADS; t++) {
        if (reader[t].buffer) free(reader[t].buffer);
    }

    return 0;
}

void print_usage(char *const argv[]) {
    fprintf(stderr,
        "Usage:\n  %s [options]\n"
        "  -p type   A=auto, C=cross, B=both (default %c)\n"
        "  -c num    channels (default %d)\n"
        "  -n num    inputs   (default %d)\n"
        "  -a num    averages before write (default %d)\n"
        "  -i file   input ('-'=stdin)\n"
        "  -o file   output('-'=stdout)\n"
        "  -s num    skip initial spectra\n"
        "  -d        debug\n",
        argv[0], prod_type, nchan, ninp, naver);
    exit(EXIT_FAILURE);
}

void parse_cmdline(int argc, char *const argv[], const char *optstring) {
    int c;
    while ((c = getopt(argc, argv, optstring)) != -1) {
        switch (c) {
            case 'c': nchan       = atoi(optarg);       break;
            case 'n': ninp        = atoi(optarg);       break;
            case 'a': naver       = atoi(optarg);       break;
            case 'p': 
                prod_type = toupper(optarg[0]); 
                if (prod_type != 'A' && prod_type != 'C' && prod_type != 'B') {
                    fprintf(stderr, "Error: Invalid product type '%c'. Must be A, C, or B.\n", prod_type);
                    exit(EXIT_FAILURE);
                }
                break;
            case 'i': infilename  = optarg;             break;
            case 'o': outfilename = optarg;             break;
            case 's': nskip       = atoi(optarg);       break;
            case 'd': debug       = 1;                  break;
            default:  print_usage(argv);
        }
    }
}

void openFiles(const char *infile, const char *outfile, int prod_type,
               FILE **fin, FILE **fout_ac, FILE **fout_cc) {
    char tmp[FILENAME_MAX];
    if (strcmp(infile, "-") == 0) {
        *fin = stdin;
    } else {
        *fin = fopen(infile, "rb");
        if (!*fin) { perror("fopen input"); exit(EXIT_FAILURE); }
    }

    *fout_ac = NULL; *fout_cc = NULL;
    if (prod_type=='A' || prod_type=='B') {
        if (strcmp(outfile, "-") == 0) *fout_ac = stdout;
        else {
            snprintf(tmp, sizeof(tmp), "%s.LACSPC", outfile);
            *fout_ac = fopen(tmp, "wb"); 
            if (!*fout_ac) { 
                perror("fopen auto"); 
                exit(EXIT_FAILURE); 
            }
        }
    }
    if (prod_type=='C' || prod_type=='B') {
        if (strcmp(outfile, "-") == 0) *fout_cc = stdout;
        else {
            snprintf(tmp, sizeof(tmp), "%s.LCCSPC", outfile);
            *fout_cc = fopen(tmp, "wb"); 
            if (!*fout_cc) { 
                perror("fopen cross"); 
                exit(EXIT_FAILURE); 
            }
        }
    }
}

float elapsed_time(struct timeval *start) {
    struct timeval now; gettimeofday(&now, NULL);
    return (now.tv_sec - start->tv_sec)*1e3f + (now.tv_usec - start->tv_usec)*1e-3f;
}

int readData(datareader_t *r, int nchan, int ninp,
             FILE *fpin, hipfftComplex **inp_buf) {
    if (!r->init) {
        r->ntoread = ninp * BYTES_PER_SAMPLE * nchan;
        r->buffer  =(unsigned char*)malloc(r->ntoread);
        if (!r->buffer) { perror("malloc"); exit(EXIT_FAILURE); }
        r->init = 1;
    }
    size_t nread = fread(r->buffer, 1, r->ntoread, fpin);
    if (nread == 0) return 1;
    if (nread < (size_t)r->ntoread)
        fprintf(stderr, "Warning: partial read %zu of %d bytes\n", nread, r->ntoread);
    for (int inp=0; inp<ninp; inp++) for (int ch=0; ch<nchan; ch++){
        size_t pos = 2*(ch*ninp+inp);
        if (pos+1>=nread) break;
        uint8_t lo=r->buffer[pos], hi=r->buffer[pos+1];
        int16_t s=(int16_t)((hi<<8)|lo);
        inp_buf[inp][ch]=make_hipFloatComplex((float)s/16.0f,0.0f);
    }
    return 0;
}

void do_FFT_fftw(cufftplan *plan, int nchan, int ninp, hipfftComplex **inp_buf, hipfftComplex **ft_buf){
    if (!plan->doneplan){
        plan->plans = (hipfftHandle*)calloc(ninp, sizeof(hipfftHandle));
        if (!plan->plans){ perror("calloc"); exit(EXIT_FAILURE);}\
        for(int i=0;i<ninp;i++) CUFFT_CHECK(hipfftPlan1d(&plan->plans[i],nchan,HIPFFT_C2C,1));
        plan->doneplan=1;
    }
    for(int i=0;i<ninp;i++) CUFFT_CHECK(hipfftExecC2C(plan->plans[i], inp_buf[i], ft_buf[i], HIPFFT_FORWARD));
}

/* accumulate correlation products */
void do_CMAC(const int nchan,const int ninp,const int prod_type, hipfftComplex **ft_buf, hipfftComplex **corr_buf) {
    int inp1,inp2,cprod=0;
    hipfftComplex *ftinp1,*ftinp2,*cbuf;
    register int chan;
    
    for(inp1=0; inp1<ninp; inp1++) {
        ftinp1 = ft_buf[inp1];
        for(inp2=inp1; inp2<ninp; inp2++) {
            ftinp2 = ft_buf[inp2];
            cbuf = corr_buf[cprod];
            if(prod_type=='B' || ((prod_type=='C' && inp1!=inp2) || (prod_type=='A' && inp1==inp2))) {
                for(chan=0;chan<nchan;chan++) {
                    cbuf[chan] += ftinp1[chan]*conjf(ftinp2[chan]);
                }
            }
            cprod++;         
        }
    }    
}

/* write out correlation products.
   Apply a normalisation factor that depends on the FFT length and the number
   of averages so the flux density is the same regardless of the spectral channel width */
void writeOutput(FILE *fout_ac, FILE *fout_cc, int ninp, int nchan, int iter, int prod_type,hipfftComplex **buf, float normaliser){

    // consider mallocing temp_buffer in main once and freeing it at the end of program execution
   float * temp_buffer = NULL; 
   temp_buffer = (float*)malloc(sizeof(float) * nchan);

   // debug
   printf("temp buffer allocated\n");

   //debug
   if (!temp_buffer) {
       perror("malloc temp_buffer");
       exit(EXIT_FAILURE);
   }
   //-----
   
   int inp1 = 0; 
   int inp2 = 0;
   int chan = 0; 
   int cprod = 0; 
    
    for(inp1=0; inp1<ninp; inp1++) {
        for (inp2=inp1; inp2<ninp; inp2++) {

            //debug
            printf("current inp1=%d inp2=%d\n", inp1, inp2);

            for (chan = 0; chan < nchan; chan++) {
                if (cprod >= ninp * (ninp + 1) / 2) {
                    fprintf(stderr, "Error: cprod out of bounds (%d >= %d)\n", cprod, ninp * (ninp + 1) / 2);
                    exit(EXIT_FAILURE);
                }
                if (!buf[cprod]) {
                    fprintf(stderr, "Error: buf[cprod] is NULL (cprod=%d)\n", cprod);
                    exit(EXIT_FAILURE);
                }
                if (chan >= nchan) {
                    fprintf(stderr, "Error: chan out of bounds (%d >= %d)\n", chan, nchan);
                    exit(EXIT_FAILURE);
                }

                buf[cprod][chan] = make_hipFloatComplex(
                    hipCrealf(buf[cprod][chan]) * normaliser,
                    hipCimagf(buf[cprod][chan]) * normaliser
                );

                if (inp1 == inp2 && (prod_type == 'A' || prod_type == 'B')) {
                    if (!temp_buffer) {
                        fprintf(stderr, "Error: temp_buffer is NULL\n");
                        exit(EXIT_FAILURE);
                    }
                    temp_buffer[chan] = hipCrealf(buf[cprod][chan]);
                }
            }

            // AC
            if(inp1==inp2 && (prod_type == 'A' || prod_type=='B')) {
                if (!fout_ac) {
                    fprintf(stderr, "Error: fout_ac is NULL\n");
                    exit(EXIT_FAILURE);
                }
                fwrite(temp_buffer, sizeof(float), nchan, fout_ac);
            }

            // CC
            if(inp1!=inp2 && (prod_type == 'C' || prod_type=='B')) {

                if (!fout_cc) {
                    fprintf(stderr, "Error: fout_cc is NULL\n");
                    exit(EXIT_FAILURE);
                }
                fwrite(buf[cprod], sizeof(hipfftComplex), nchan, fout_cc);
            }

            /* reset the correlation products to zero */

            if (!buf[cprod]) {
                fprintf(stderr, "Error: buf[cprod] is NULL (cprod=%d)\n", cprod);
                exit(EXIT_FAILURE);
            }
            
            memset(buf[cprod], '\0', (nchan) * sizeof(hipfftComplex));
            cprod++;
        }
    }
    if (temp_buffer!=NULL) free(temp_buffer);
}
