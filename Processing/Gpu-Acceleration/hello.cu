#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


__global__ void printing(char * message){
printf("%s", message);
}


int main ()
{
char * message = new char[100];
printf("Type the message:/n ");
scanf("%99s", message);


char * d_message; //gpu v
hipMalloc((void**)&d_message, 100 * sizeof(char));

//then you have to copy it to the gpu 
hipMemcpy(d_message, message, 100 * sizeof(char), hipMemcpyHostToDevice);


//then you have to launch the kernel 
printing<<<1,1>>>(message);
//this ensures the gpu finishes processing 
hipDeviceSynchronize();

//then finally free all of the allocated mema

hipFree(d_message);

return 0;
}
